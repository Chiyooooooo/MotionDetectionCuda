#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <cstdio>
#include <cmath>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>

// Macro pour vérifier les erreurs CUDA, utile pour le débugging
#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)

template <typename T>
void check(T err, const char* const func, const char* const file, const int line) {
    if (err != hipSuccess) {
        std::fprintf(stderr, "Erreur Runtime CUDA à : %s: %d\n", file, line);
        std::fprintf(stderr, "%s %s\n", hipGetErrorString(err), func);
        std::exit(EXIT_FAILURE);
    }
}

// Struct pour la couleur RGB uint8 
struct rgb {
    uint8_t r, g, b;
};

struct lab {
    float l, a, b;
};

__device__ float correction_gamma(float color) {
    return (color <= 0.04045f) ? color / 12.92f : powf((color + 0.055f) / 1.055f, 2.4f);
}

__device__ float f(float t) {
    return (t > 0.008856f) ? powf(t, 1.0f / 3.0f) : (7.787f * t) + (16.0f / 116.0f);
}

// Conversion RGB vers LAB (je sais pas si les formules sont correctes)
__device__ lab RGBtoLAB(const rgb& rgb) {
    float r = correction_gamma(rgb.r / 255.0f);
    float g = correction_gamma(rgb.g / 255.0f);
    float b = correction_gamma(rgb.b / 255.0f);

    // Conversion en espace de couleur XYZ
    float X = r * 0.4124564f + g * 0.3575761f + b * 0.1804375f;
    float Y = r * 0.2126729f + g * 0.7151522f + b * 0.0721750f;
    float Z = r * 0.0193339f + g * 0.1191920f + b * 0.9503041f;

    // Conversion en espace de couleur LAB
    float fx = f(X / 95.047f);
    float fy = f(Y / 100.0f);
    float fz = f(Z / 108.883f);

    return { (116.0f * fy) - 16.0f, 500.0f * (fx - fy), 200.0f * (fy - fz) };
}

// Kernel pour convertir une frame RGB en frame LAB 
__global__ void RgbToLabKernel(const rgb* rgbFrame, lab* labFrame, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * width + x;

    if (x < width && y < height) {
        labFrame[idx] = RGBtoLAB(rgbFrame[idx]);
    }
}

// Fonction device pour calculer la différence de couleur 
__device__ float deltaE(const lab& pixel1, const lab& pixel2) {
    return sqrtf(
            (pixel1.l - pixel2.l) * (pixel1.l - pixel2.l) +
            (pixel1.a - pixel2.a) * (pixel1.a - pixel2.a) +
            (pixel1.b - pixel2.b) * (pixel1.b - pixel2.b)
    );
}

__global__ void ComputeResidualKernel(const lab* img1, const lab* img2, float* residual, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * width + x;

    if (x < width && y < height) {
        residual[idx] = deltaE(img1[idx], img2[idx]);
    }
}

// Kernel pour normaliser l'image résiduelle 
__global__ void normalizeResidualKernel(const float* residual, uint8_t* normalizedResidual, float maxResidual, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * width + x;

    if (x < width && y < height) {
        normalizedResidual[idx] = static_cast<uint8_t>((residual[idx] / maxResidual) * 255.0f);
    }
}

// erosion
__global__ void erodeKernel(const uint8_t* src, uint8_t* dst, int width, int height, int radius) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * width + x;

    if (x < width && y < height) {
        uint8_t minPixel = 255;
        for (int dy = -radius; dy <= radius; ++dy) {
            for (int dx = -radius; dx <= radius; ++dx) {
                int nx = x + dx;
                int ny = y + dy;
                if (nx >= 0 && nx < width && ny >= 0 && ny < height) {
                    uint8_t currentPixel = src[ny * width + nx];
                    if (currentPixel < minPixel) {
                        minPixel = currentPixel;
                    }
                }
            }
        }
        dst[idx] = minPixel;
    }
}

// Kernel pour la dilatation morphologique (pareil ici)
__global__ void dilateKernel(const uint8_t* src, uint8_t* dst, int width, int height, int radius) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * width + x;

    if (x < width && y < height) {
        uint8_t maxPixel = 0;
        for (int dy = -radius; dy <= radius; ++dy) {
            for (int dx = -radius; dx <= radius; ++dx) {
                int nx = x + dx;
                int ny = y + dy;
                if (nx >= 0 && nx < width && ny >= 0 && ny < height) {
                    uint8_t currentPixel = src[ny * width + nx];
                    if (currentPixel > maxPixel) {
                        maxPixel = currentPixel;
                    }
                }
            }
        }
        dst[idx] = maxPixel;
    }
}

__global__ void hysteresisThresholdKernel(const uint8_t* src, uint8_t* dst, bool* strong_edges, int width, int height, int seuil_min, int seuil_max) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * width + x;

    if (x < width && y < height) {
        uint8_t pixel = src[idx];
        if (pixel > seuil_max) {
            strong_edges[idx] = true;
            dst[idx] = 255;
        } else {
            dst[idx] = 0;
        }
    }
}


__global__ void StrongEdgesPropagationKernel(const uint8_t* src, uint8_t* dst, bool* strong_edges, int width, int height, int seuil_min) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * width + x;

    if (x < width && y < height && strong_edges[idx]) {
        for (int dy = -1; dy <= 1; ++dy) {
            for (int dx = -1; dx <= 1; ++dx) {
                int nx = x + dx;
                int ny = y + dy;
                if (nx >= 0 && nx < width && ny >= 0 && ny < height) {
                    int n_idx = ny * width + nx;
                    uint8_t neighbor_pixel = src[n_idx];
                    if (neighbor_pixel >= seuil_min && !strong_edges[n_idx]) {
                        strong_edges[n_idx] = true;
                        dst[n_idx] = 255;
                    }
                }
            }
        }
    }
}

// a fix lae rgb
__global__ void putMaskKernel(const rgb* input, const uint8_t* mask, rgb* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * width + x;

    if (x < width && y < height) {
        rgb red = {255, 0, 0};
        rgb pixel = input[idx];
        uint8_t pixel_mask = mask[idx];
        rgb pixel_output;

        if (pixel_mask > 0) {
            // //
            pixel_output.r = static_cast<uint8_t>(min(255.0f, pixel.r * 0.5f + red.r * 0.5f));
            pixel_output.g = static_cast<uint8_t>(min(255.0f, pixel.g * 0.5f + red.g * 0.5f));
            pixel_output.b = static_cast<uint8_t>(min(255.0f, pixel.b * 0.5f + red.b * 0.5f));
        } else {
            pixel_output = pixel;
        }

        output[idx] = pixel_output;
    }
}

// Fonction pour calculer et normalisefaut laa check)
void computeAndNormalizeResidual(const lab* img1, const lab* img2, float* residual, uint8_t* normalizedResidual, int width, int height, hipStream_t stream) {
    int size = width * height;
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    ComputeResidualKernel<<<gridSize, blockSize, 0, stream>>>(img1, img2, residual, width, height);

    thrust::device_ptr<float> dev_ptr_residual(residual);
    float maxResidual = *thrust::max_element(thrust::device, dev_ptr_residual, dev_ptr_residual + size);

    normalizeResidualKernel<<<gridSize, blockSize, 0, stream>>>(residual, normalizedResidual, maxResidual, width, height);
}

// Fonction pour l'ouverture morphologique (érosion suivie de dilatation) sans fermutre, le plus simple pour optimiser le temps de calcul
void morphologicalOpening(const uint8_t* src, uint8_t* dst, int width, int height, hipStream_t stream) {
    int radius = 3; // Rayon ste a 3 comme dans le pdf 
    uint8_t* d_temp;
    size_t size = width * height * sizeof(uint8_t);

    hipMalloc(&d_temp, size);

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    erodeKernel<<<gridSize, blockSize, 0, stream>>>(src, d_temp, width, height, radius);
    dilateKernel<<<gridSize, blockSize, 0, stream>>>(d_temp, dst, width, height, radius);

    hipFree(d_temp);
}

// Fonction pour l''hystérésis
void hysteresisThreshold(const uint8_t* src, uint8_t* dst, int width, int height, int seuil_min, int seuil_max, hipStream_t stream) {
    size_t bool_size = width * height * sizeof(bool);
    bool* d_strong_edges;

    hipMalloc(&d_strong_edges, bool_size);
    hipMemset(d_strong_edges, 0, bool_size);

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    hysteresisThresholdKernel<<<gridSize, blockSize, 0, stream>>>(src, dst, d_strong_edges, width, height, seuil_min, seuil_max);
    StrongEdgesPropagationKernel<<<gridSize, blockSize, 0, stream>>>(src, dst, d_strong_edges, width, height, seuil_min);

    hipFree(d_strong_edges);
}

// Fonction pour appliquer un masque à l'image
void putMask(const rgb* input, const uint8_t* mask, rgb* output, int width, int height, hipStream_t stream) {
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    putMaskKernel<<<gridSize, blockSize, 0, stream>>>(input, mask, output, width, height);
}

// Struct pour contenir les données du modèle de fond
struct BackgroundModel {
    rgb* data = nullptr;
    int width = 0;
    int height = 0;
    int stride = 0;
    bool is_initialized = false;

    ~BackgroundModel() {
        delete[] data;
    }
};

BackgroundModel bg_model;

// Fonction externe pour filtrer l'image
extern "C" {
void filter_impl(uint8_t* buffer, int width, int height, int stride, int pixel_stride) {
    static int frame_count = 0;
    static bool first_call = true;
    static rgb *d_frame_rgb = nullptr, *d_bg_rgb = nullptr, *d_output = nullptr;
    static lab *d_frame_lab = nullptr, *d_bg_lab = nullptr;
    static float *d_residual = nullptr;
    static uint8_t *d_residual_normalized = nullptr, *d_opened = nullptr, *d_hysteresis = nullptr;

    frame_count++;
    if (!bg_model.is_initialized) {
        bg_model.data = new rgb[width * height];
        bg_model.width = width;
        bg_model.height = height;
        bg_model.stride = stride;
        bg_model.is_initialized = true;

        for (int y = 0; y < height; ++y) {
            memcpy(bg_model.data + y * width, buffer + y * stride, width * sizeof(rgb));
        }
        return;
    }

    size_t rgb_size = width * height * sizeof(rgb);
    size_t lab_size = width * height * sizeof(lab);
    size_t float_size = width * height * sizeof(float);
    size_t uint8_size = width * height * sizeof(uint8_t);

    if (first_call) {
        hipMalloc(&d_frame_rgb, rgb_size);
        hipMalloc(&d_bg_rgb, rgb_size);
        hipMalloc(&d_frame_lab, lab_size);
        hipMalloc(&d_bg_lab, lab_size);
        hipMalloc(&d_residual, float_size);
        hipMalloc(&d_residual_normalized, uint8_size);
        hipMalloc(&d_opened, uint8_size);
        hipMalloc(&d_hysteresis, uint8_size);
        hipMalloc(&d_output, rgb_size);
        first_call = false;
    }

    hipMemcpy(d_frame_rgb, buffer, rgb_size, hipMemcpyHostToDevice);
    hipMemcpy(d_bg_rgb, bg_model.data, rgb_size, hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    hipStream_t stream;
    hipStreamCreate(&stream);

    RgbToLabKernel<<<gridSize, blockSize, 0, stream>>>(d_frame_rgb, d_frame_lab, width, height);
    RgbToLabKernel<<<gridSize, blockSize, 0, stream>>>(d_bg_rgb, d_bg_lab, width, height);

    computeAndNormalizeResidual(d_bg_lab, d_frame_lab, d_residual, d_residual_normalized, width, height, stream);
    morphologicalOpening(d_residual_normalized, d_opened, width, height, stream);
    hysteresisThreshold(d_opened, d_hysteresis, width, height, 4, 30, stream);
    putMask(d_frame_rgb, d_hysteresis, d_output, width, height, stream);

    hipMemcpy(buffer, d_output, rgb_size, hipMemcpyDeviceToHost);

    hipStreamDestroy(stream);
}
}
